
#include <hip/hip_runtime.h>

inline __global__ void KernelTest(int * a, int * b, int * res, int size)
{
	// Calcul de l'indice du tableau
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	res[x] = a[x] + b[x];
}

extern "C"
void runKernel(int * a, int * b, int * res, int size)
{
	dim3 dimBlock(64, 1, 1);
    dim3 dimGrid(size / dimBlock.x, 1, 1);
    
	KernelTest<<< dimGrid, dimBlock, 0 >>>( a, b, res, size);
}
